#include <cstdio>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d (%d): %s\n", __FILE__, __LINE__,    \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

__global__ void kernelA() { printf("kernelA running on stream1\n"); }

__global__ void kernelB() {
  printf("kernelB running on stream2 AFTER kernelA\n");
}

int main() {
  hipStream_t stream1, stream2;
  CHECK_CUDA(hipStreamCreate(&stream1));
  CHECK_CUDA(hipStreamCreate(&stream2));

  hipEvent_t extEvent;
  CHECK_CUDA(hipEventCreateWithFlags(&extEvent, hipEventDisableTiming));

  hipGraph_t graph;
  CHECK_CUDA(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));

  // kernelA on stream1
  kernelA<<<1, 1, 0, stream1>>>();

  CHECK_CUDA(
       hipEventRecord(extEvent, stream1));
  // stream2 waits on extEvent (from stream1) inside capture
  CHECK_CUDA(hipStreamWaitEvent(stream1, extEvent));

  // kernelB on stream2
  kernelB<<<1, 1, 0, stream2>>>();

  CHECK_CUDA(hipStreamEndCapture(stream1, &graph));

  // Instantiate and launch graph
  hipGraphExec_t graphExec;
  CHECK_CUDA(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  CHECK_CUDA(hipGraphLaunch(graphExec, stream1));
  CHECK_CUDA(hipStreamSynchronize(stream1));

  // Cleanup
  CHECK_CUDA(hipGraphExecDestroy(graphExec));
  CHECK_CUDA(hipGraphDestroy(graph));
  CHECK_CUDA(hipEventDestroy(extEvent));
  CHECK_CUDA(hipStreamDestroy(stream1));
  CHECK_CUDA(hipStreamDestroy(stream2));

  return 0;
}
