#include <cstdio>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d (%d): %s\n", __FILE__, __LINE__,    \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

__global__ void kernelA() { printf("kernelA running on stream1\n"); }

__global__ void kernelB() {
  printf("kernelB running on stream2 AFTER kernelA\n");
}

int main() {
  hipStream_t stream1, stream2;
  CHECK_CUDA(hipStreamCreate(&stream1));
  CHECK_CUDA(hipStreamCreate(&stream2));

  hipEvent_t extEvent;
  CHECK_CUDA(hipEventCreateWithFlags(&extEvent, hipEventDisableTiming));

  hipGraph_t graph;
  CHECK_CUDA(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));

  // kernelA on stream1
  kernelA<<<1, 1, 0, stream1>>>();

  CHECK_CUDA(
      hipEventRecordWithFlags(extEvent, stream1, hipEventRecordExternal));
  // if you use stream2 it fails: (401): the operation cannot be performed in
  // the present state
  CHECK_CUDA(hipStreamWaitEvent(stream1, extEvent, cudaEventWaitExternal));

  // kernelB on stream2
  kernelB<<<1, 1, 0, stream2>>>();

  CHECK_CUDA(hipStreamEndCapture(stream1, &graph));

  // Instantiate and launch graph
  hipGraphExec_t graphExec;
  CHECK_CUDA(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  CHECK_CUDA(hipGraphLaunch(graphExec, stream1));
  CHECK_CUDA(hipStreamSynchronize(stream1));

  // Cleanup
  CHECK_CUDA(hipGraphExecDestroy(graphExec));
  CHECK_CUDA(hipGraphDestroy(graph));
  CHECK_CUDA(hipEventDestroy(extEvent));
  CHECK_CUDA(hipStreamDestroy(stream1));
  CHECK_CUDA(hipStreamDestroy(stream2));

  return 0;
}
